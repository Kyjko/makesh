#include "hip/hip_runtime.h"
/*
    Written by Kyjko (Bognár Miklós)
    CUDA - vector operations on GPU - with SDL
*/

#include <stdio.h>
#include <time.h>
#include <memory.h>
#include <stdlib.h>
#include <SDL.h>

#undef main

#define N 20
#define W 1920
#define H 1080

short quit = 0;

enum Operators {
    ADD,
    SUB,
    MUL
};

typedef struct context {
    SDL_Window* w;
    SDL_Renderer* r;
} context;

__global__ void kernel(enum Operators type, float* a, float* b, float* c, unsigned long n) {
    unsigned long idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(idx > n)
        return;
    
        switch(type) {
        case ADD:
            c[idx] = a[idx] + b[idx];
        case SUB:
            c[idx] = a[idx] - b[idx];
        case MUL:
            c[idx] = a[idx] * b[idx];
        default: 
            return;
        
    }
}

__host__ void display_vectors(enum Operators type, float* a, float* b, float* c, unsigned long n) {
    for(int i = 0; i < N; i++) {
        printf("%d. : %f %s %f = %f\n", i, a[i], 
            type == ADD ? "+" : type == SUB ? "-" : type == MUL ? "*" : "?", b[i], c[i]);
    }
}

errno_t init_sdl(context* ctx) {
    if(SDL_Init(SDL_INIT_VIDEO) < 0)
        return -1;
    
    ctx->w = SDL_CreateWindow("Numpu", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, W, H, SDL_WINDOW_SHOWN);
    if(ctx->w == NULL)
        return -2;
    ctx->r = SDL_CreateRenderer(ctx->w, -1, SDL_RENDERER_ACCELERATED);
    if(ctx->r == NULL)
        return -3;
    

    return 0;

}

void render(context* ctx) {
    
}

void eventloop(context* ctx) {
    while(quit != 1) {
        SDL_Event e;
        while(SDL_PollEvent(&e) != NULL) {
            switch(e.type) {
                case SDL_QUIT:
                    quit = 1;
                    break;
            }
        }

        render(ctx);
    }

    SDL_DestroyRenderer(ctx->r);
    SDL_DestroyWindow(ctx->w);
    ctx->w = NULL;
    ctx->r = NULL;
    SDL_Quit();
}

int main(int argc, char** argv) {
    srand((unsigned)time(NULL));
    
    context ctx;
    memset(&ctx, 0, sizeof(ctx));

    //init sdl
    if(init_sdl(&ctx) < 0) {
        perror("cannot initialize SDL!");    
        return -1;
    }

    eventloop(&ctx);

    float* a = (float*)malloc(sizeof(float)*N);
    float* b = (float*)malloc(sizeof(float)*N);
    float* c = (float*)malloc(sizeof(float)*N);

    memset(c, 0.0, sizeof(float)*N);
    
    for(int i = 0; i < N; i++) {
        a[i] = (rand()/(float)RAND_MAX)*100;
        b[i] = (rand()/(float)RAND_MAX)*100;
    }

    float *d_a, *d_b, *d_c;

    hipMalloc((void**)&d_a, sizeof(float)*N);
    hipMalloc((void**)&d_b, sizeof(float)*N);
    hipMalloc((void**)&d_c, sizeof(float)*N);

    hipMemcpy(d_a, a, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, sizeof(float)*N, hipMemcpyHostToDevice);

    kernel<<<100, 100>>> (MUL, d_a, d_b, d_c, N);

    hipMemcpy(a, d_a, sizeof(float)*N, hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, sizeof(float)*N, hipMemcpyDeviceToHost);
    hipMemcpy(c, d_c, sizeof(float)*N, hipMemcpyDeviceToHost);

    display_vectors(MUL, a, b, c, N);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);
    a = NULL;
    b = NULL;
    c = NULL;

    return 0;
}
